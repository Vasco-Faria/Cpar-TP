#include "hip/hip_runtime.h"
#include "fluid_solver.h"
#include <cmath>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))
#define SWAP(x0, x) { float *tmp = x0; x0 = x; x = tmp; }
#define MAX(a, b) (((a) > (b)) ? (a) : (b)) 

static int size;

// variaveis kernels
// set_bnd 
float* d_x = nullptr;
// lin_solve
float* d_xx = nullptr;
float* d_x0 = nullptr;
float* d_max_change = nullptr;
// advect
float* d_d = nullptr;
float* d_d0 = nullptr;
float* d_u = nullptr;
float* d_v = nullptr;
float* d_w = nullptr;
// project
float* d_uu = nullptr;
float* d_vv = nullptr;
float* d_ww = nullptr;
float* d_pp = nullptr;
float* d_div = nullptr;

void init_cuda_mallocs(int M, int N, int O) {
    size = (M + 2) * (N + 2) * (O + 2) * sizeof(float);
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_xx, size);
    hipMalloc((void**)&d_x0, size);
    hipMalloc((void**)&d_max_change, size);
    hipMalloc((void**)&d_d, size);
    hipMalloc((void**)&d_d0, size);
    hipMalloc((void**)&d_u, size);
    hipMalloc((void**)&d_v, size);
    hipMalloc((void**)&d_w, size);
    hipMalloc((void**)&d_uu, size);
    hipMalloc((void**)&d_vv, size);
    hipMalloc((void**)&d_ww, size);
    hipMalloc((void**)&d_pp, size);
    hipMalloc((void**)&d_div, size);
}

void free_cuda_mallocs() {
    hipFree(d_x);
    hipFree(d_xx);
    hipFree(d_x0);
    hipFree(d_max_change);
    hipFree(d_d);
    hipFree(d_d0);
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_w);
    hipFree(d_uu);
    hipFree(d_vv);
    hipFree(d_ww);
    hipFree(d_pp);
    hipFree(d_div);
}

void add_source(int M, int N, int O, float *x, float *s, float dt) {
    int size_ = (M + 2) * (N + 2) * (O + 2);
    for (int i = 0; i < size_; i++) {
        x[i] += dt * s[i];
    }
}

__global__ void set_bnd_kernel(int M, int N, int O, int b, float* x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= 1 && i <= M && j >= 1 && j <= N && k >= 1 && k <= O) {
        // k = 0 and k = O+1
        if (k == 0) x[IX(i, j, 0)] = (b == 3) ? -x[IX(i, j, 1)] : x[IX(i, j, 1)];
        if (k == O + 1) x[IX(i, j, O + 1)] = (b == 3) ? -x[IX(i, j, O)] : x[IX(i, j, O)];

        // i = 0 and i = M+1
        if (i == 0) x[IX(0, j, k)] = (b == 1) ? -x[IX(1, j, k)] : x[IX(1, j, k)];
        if (i == M + 1) x[IX(M + 1, j, k)] = (b == 1) ? -x[IX(M, j, k)] : x[IX(M, j, k)];

        // j = 0 and j = N+1
        if (j == 0) x[IX(i, 0, k)] = (b == 2) ? -x[IX(i, 1, k)] : x[IX(i, 1, k)];
        if (j == N + 1) x[IX(i, N + 1, k)] = (b == 2) ? -x[IX(i, N, k)] : x[IX(i, N, k)];
    }
    
    if (i == 0 && j == 0 && k == 0) 
        x[IX(0, 0, 0)] = 0.33f * (x[IX(1, 0, 0)] + x[IX(0, 1, 0)] + x[IX(0, 0, 1)]);
    if (i == M + 1 && j == 0 && k == 0) 
        x[IX(M + 1, 0, 0)] = 0.33f * (x[IX(M, 0, 0)] + x[IX(M + 1, 1, 0)] + x[IX(M + 1, 0, 1)]);
    if (i == 0 && j == N + 1 && k == 0) 
        x[IX(0, N + 1, 0)] = 0.33f * (x[IX(1, N + 1, 0)] + x[IX(0, N, 0)] + x[IX(0, N + 1, 1)]);
    if (i == M + 1 && j == N + 1 && k == 0) 
        x[IX(M + 1, N + 1, 0)] = 0.33f * (x[IX(M, N + 1, 0)] + x[IX(M + 1, N, 0)] + x[IX(M + 1, N + 1, 1)]);
}

void set_bnd(int M, int N, int O, int b, float *x) {
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, d_x);
    hipDeviceSynchronize();

    hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost);
}

__global__ void lin_solve_red_kernel(int M, int N, int O, int b, float* x, const float* x0, float a, float inv_c, float* max_change) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        if ((i + j + k) % 2 == 1) { 
            int index = IX(i, j, k);
            float old_x = x[index];
            x[index] = (x0[index] +
                        a * (x[IX(i - 1, j, k)] + x[IX(i + 1, j, k)] +
                             x[IX(i, j - 1, k)] + x[IX(i, j + 1, k)] +
                             x[IX(i, j, k - 1)] + x[IX(i, j, k + 1)])) * inv_c;
            float change = fabsf(x[index] - old_x);
            if (change > *max_change) *max_change = change;
        }
    }
}

__global__ void lin_solve_black_kernel(int M, int N, int O, int b, float* x, const float* x0, float a, float inv_c, float* max_change) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        if ((i + j + k) % 2 == 0) { 
            int index = IX(i, j, k);
            float old_x = x[index];
            x[index] = (x0[index] +
                        a * (x[IX(i - 1, j, k)] + x[IX(i + 1, j, k)] +
                             x[IX(i, j - 1, k)] + x[IX(i, j + 1, k)] +
                             x[IX(i, j, k - 1)] + x[IX(i, j, k + 1)])) * inv_c;
            float change = fabsf(x[index] - old_x);
            if (change > *max_change) *max_change = change;
        }
    }
}

void lin_solve(int M, int N, int O, int b, float* x, const float* x0, float a, float c) {
    float tol = 1e-7f;
    float max_change;

    hipMemcpy(d_xx, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_x0, x0, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);

    float inv_c = 1.0f / c;
    int iterations = 0;

    do {
        max_change = 0.0f;
        hipMemcpy(d_max_change, &max_change, sizeof(float), hipMemcpyHostToDevice);

        lin_solve_red_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, d_xx, d_x0, a, inv_c, d_max_change);
        hipDeviceSynchronize();

        lin_solve_black_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, d_xx, d_x0, a, inv_c, d_max_change);
        hipDeviceSynchronize();

        hipMemcpy(&max_change, d_max_change, sizeof(float), hipMemcpyDeviceToHost);

        set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, d_xx);
        hipDeviceSynchronize();

    } while (max_change > tol && ++iterations < 20);

    hipMemcpy(x, d_xx, size, hipMemcpyDeviceToHost);
}

void diffuse(int M, int N, int O, int b, float *x, float *x0, float diff, float dt) {
    int max = MAX(M, MAX(N, O));
    float a = dt * diff * max * max;
    lin_solve(M, N, O, b, x, x0, a, 1 + 6 * a);
}

__global__ void advect_kernel(int M, int N, int O, int b, float *d, float *d0, float *u, float *v, float *w, float dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1; // +1 para evitar bordas
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i > M || j > N || k > O) return;

    int index = IX(i, j, k);
    float dtX = dt * M, dtY = dt * N, dtZ = dt * O;

    float u_val = u[index], v_val = v[index], w_val = w[index];
    float x = i - dtX * u_val, y = j - dtY * v_val, z = k - dtZ * w_val;

    x = (x < 0.5f) ? 0.5f : (x > M + 0.5f) ? M + 0.5f : x;
    y = (y < 0.5f) ? 0.5f : (y > N + 0.5f) ? N + 0.5f : y;
    z = (z < 0.5f) ? 0.5f : (z > O + 0.5f) ? O + 0.5f : z;

    int i0 = (int)x, i1 = i0 + 1, j0 = (int)y, j1 = j0 + 1, k0 = (int)z, k1 = k0 + 1;
    float s1 = x - i0, s0 = 1 - s1, t1 = y - j0, t0 = 1 - t1, u1 = z - k0, u0 = 1 - u1;

    d[index] = 
        s0 * (t0 * (u0 * d0[IX(i0, j0, k0)] + u1 * d0[IX(i0, j0, k1)]) + 
              t1 * (u0 * d0[IX(i0, j1, k0)] + u1 * d0[IX(i0, j1, k1)])) +
        s1 * (t0 * (u0 * d0[IX(i1, j0, k0)] + u1 * d0[IX(i1, j0, k1)]) + 
              t1 * (u0 * d0[IX(i1, j1, k0)] + u1 * d0[IX(i1, j1, k1)]));
}

void advect(int M, int N, int O, int b, float *d, float *d0, float *u, float *v, float *w, float dt) {
    hipMemcpy(d_d, d, size, hipMemcpyHostToDevice);
    hipMemcpy(d_d0, d0, size, hipMemcpyHostToDevice);
    hipMemcpy(d_u, u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, size, hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);

    advect_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, d_d, d_d0, d_u, d_v, d_w, dt);
    hipDeviceSynchronize();

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, b, d_d);
    hipDeviceSynchronize();

    hipMemcpy(d, d_d, size, hipMemcpyDeviceToHost);
}

__global__ void compute_div_and_init_p(int M, int N, int O, float *u, float *v, float *w, float *p, float *div, float inverso_MNO) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        int index = IX(i, j, k);
        div[index] = -0.5f * (
            u[IX(i + 1, j, k)] - u[IX(i - 1, j, k)] +
            v[IX(i, j + 1, k)] - v[IX(i, j - 1, k)] +
            w[IX(i, j, k + 1)] - w[IX(i, j, k - 1)]
        ) * inverso_MNO;

        p[index] = 0.0f;
    }
}

__global__ void update_velocities(int M, int N, int O, float *u, float *v, float *w, float *p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i <= M && j <= N && k <= O) {
        int index = IX(i, j, k);
        u[index] -= 0.5f * (p[IX(i + 1, j, k)] - p[IX(i - 1, j, k)]);
        v[index] -= 0.5f * (p[IX(i, j + 1, k)] - p[IX(i, j - 1, k)]);
        w[index] -= 0.5f * (p[IX(i, j, k + 1)] - p[IX(i, j, k - 1)]);
    }
}

void project(int M, int N, int O, float *u, float *v, float *w, float *p, float *div) {
    float inverso_MNO = 1.0f / (MAX(M, MAX(N, O)));

    hipMemcpy(d_uu, u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vv, v, size, hipMemcpyHostToDevice);
    hipMemcpy(d_ww, w, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pp, p, size, hipMemcpyHostToDevice);
    hipMemcpy(d_div, div, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (O + threadsPerBlock.z - 1) / threadsPerBlock.z);

    compute_div_and_init_p<<<numBlocks, threadsPerBlock>>>(M, N, O, d_uu, d_vv, d_ww, d_pp, d_div, inverso_MNO);
    hipDeviceSynchronize();

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 0, d_div);
    hipDeviceSynchronize();

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 0, d_pp);
    hipDeviceSynchronize();

    lin_solve(M, N, O, 0, d_pp, d_div, 1 ,6);

    update_velocities<<<numBlocks, threadsPerBlock>>>(M, N, O, d_uu, d_vv, d_ww, d_pp);
    hipDeviceSynchronize();

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 1, d_uu);
    hipDeviceSynchronize();

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 2, d_vv);
    hipDeviceSynchronize();

    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O, 3, d_ww);
    hipDeviceSynchronize();

    hipMemcpy(u, d_uu, size, hipMemcpyDeviceToHost);
    hipMemcpy(v, d_vv, size, hipMemcpyDeviceToHost);
    hipMemcpy(w, d_ww, size, hipMemcpyDeviceToHost);
    hipMemcpy(p, d_pp, size, hipMemcpyDeviceToHost);
    hipMemcpy(div, d_div, size, hipMemcpyDeviceToHost);
}

void dens_step(int M, int N, int O, float *x, float *x0, float *u, float *v, float *w, float diff, float dt) {
    add_source(M, N, O, x, x0, dt);
    SWAP(x0, x);
    diffuse(M, N, O, 0, x, x0, diff, dt);
    SWAP(x0, x);
    advect(M, N, O, 0, x, x0, u, v, w, dt);
    free_cuda_mallocs();
}

void vel_step(int M, int N, int O, float *u, float *v, float *w, float *u0, float *v0, float *w0, float visc, float dt) {
    init_cuda_mallocs(M, N, O);
    add_source(M, N, O, u, u0, dt);
    add_source(M, N, O, v, v0, dt);
    add_source(M, N, O, w, w0, dt);
    SWAP(u0, u);
    diffuse(M, N, O, 1, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(M, N, O, 2, v, v0, visc, dt);
    SWAP(w0, w);
    diffuse(M, N, O, 3, w, w0, visc, dt);
    project(M, N, O, u, v, w, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    SWAP(w0, w);
    advect(M, N, O, 1, u, u0, u0, v0, w0, dt);
    advect(M, N, O, 2, v, v0, u0, v0, w0, dt);
    advect(M, N, O, 3, w, w0, u0, v0, w0, dt);
    project(M, N, O, u, v, w, u0, v0);
}
